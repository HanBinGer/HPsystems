﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t printWithCuda(unsigned int size);

__global__ void printKernel()
{
    int i = threadIdx.x;
    printf("Message from kernel %d \n", i);
}

int main()
{
    const int size = 5;

    // Add vectors in parallel.
    hipError_t cudaStatus = printWithCuda(size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "printWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t printWithCuda(unsigned int size)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    // Launch a kernel on the GPU with one thread for each element.
    printKernel<<<1, size>>>();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "printKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching printKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    return cudaStatus;
}
